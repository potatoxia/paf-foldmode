#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>

#include "paf_process.cuh"
#include "process.cuh"
#include "cudautil.cuh"

int main(int argc, char *argv[])
{
  int arg;
  conf_t conf;

  /* Setup log interface */
  conf.log = multilog_open("paf_process", 0);
  conf.fp_log = fopen("paf_process.log", "ab+");
  multilog_add(conf.log, conf.fp_log);
  multilog(conf.log, LOG_INFO, "START PAF_PROCESS\n");
  
  /* Initial part */  
  while((arg=getopt(argc,argv,"c:o:i:d:s:h:n:p:r:g:f:b:")) != -1)
    {
      switch(arg)
	{	  
	case 'h':	  	  
	  sscanf(optarg, "%s", conf.hfname);
	  break;

	case 'c':
	  sscanf(optarg, "%lf", &conf.rbufin_ndfstp);
	  break;
	  
	case 's':
	  sscanf(optarg, "%d", &conf.sod);
	  break;
	  
	case 'o':	  
	  if (sscanf (optarg, "%x", &conf.key_out) != 1)
	    {
	      multilog (conf.log, LOG_ERR, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  
	case 'i':	  
	  if (sscanf (optarg, "%x", &conf.key_in) != 1)
	    {
	      multilog (conf.log, LOG_ERR, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  
	case 'd':
	  sscanf(optarg, "%d", &conf.device_id);
	  break;

	case 'n':
	  sscanf(optarg, "%d", &conf.nstream);
	  break;
	  
	case 'p':
	  sscanf(optarg, "%d", &conf.stream_ndfstp);
	  break;
	  
	case 'g':
	  sscanf(optarg, "%d", &conf.debug);
	  break;
	  
	case 'f':
	  sscanf(optarg, "%s", conf.dir);
	  break;
	  
	case 'b':
	  sscanf(optarg, "%d", &conf.nrun_blk);
	  break;	  
	}
    }
  
#ifdef DEBUG
  struct timespec start, stop;
  double elapsed_time;
  clock_gettime(CLOCK_REALTIME, &start);
#endif
  init_process(&conf);
#ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &stop);
      elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1000000000.0L;
      fprintf(stdout, "elapsed time for processing prepare is %f s\n\n\n\n\n", elapsed_time);
#endif
      
  /* Check on-board gpus */
//#ifdef DEBUG
//  int deviceCount, device;
//  struct hipDeviceProp_t properties;
//  CudaSafeCall(hipGetDeviceCount(&deviceCount));
//  fprintf(stdout, "Number of devices %d\n", deviceCount);
//  for(device = 0; device < deviceCount; ++device)
//    {
//      hipGetDeviceProperties(&properties, device);
//      if (properties.major != 9999) /* 9999 means emulation only */
//  	{
//  	  printf("multiProcessorCount %d\n",properties.multiProcessorCount);
//  	  printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
//  	  printf("pciDeviceID %d\n",properties.pciDeviceID);
//  	  printf("pciBusID %d\n",properties.pciBusID);
//  	}
//    }
//#endif
  
  /* Play with data */
#ifdef DEBUG
  clock_gettime(CLOCK_REALTIME, &start);
#endif
  if(do_process(conf))
    {
      multilog (conf.log, LOG_ERR, "Can not finish the process, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Can not finish the process, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  #ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &stop);
      elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1000000000.0L;
      fprintf(stdout, "elapsed time for data processing is %f s\n", elapsed_time);
#endif

  destroy_process(conf);

  /* Destory log interface */
  multilog(conf.log, LOG_INFO, "FINISH PAF_PROCESS\n\n");
  multilog_close(conf.log);
  fclose(conf.fp_log);
  
  return EXIT_SUCCESS;
}